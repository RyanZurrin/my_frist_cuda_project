#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "cuda_common.cuh"
#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cstring>

__global__ void sum_array_gpu(int * a, int * b, int * c, int size)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
}

void sum_array_cpu(int* a, int* b, int* c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}


int main()
{
	int size = 1 << 25;
	int block_size = 1024;
	hipError_t error;

	size_t NO_BYTES = size * sizeof(int);

	int* h_a, * h_b, * gpu_results, *cpu_results;

	h_a = (int*)malloc(NO_BYTES);
	h_b = (int*)malloc(NO_BYTES);
	gpu_results = (int*)malloc(NO_BYTES);
	cpu_results = (int*)malloc(NO_BYTES);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		h_a[i] = (int)(rand() & 0xff);
	}
	for (int i = 0; i < size; i++)
	{
		h_b[i] = (int)(rand() & 0xff);
	}
	

	memset(gpu_results, 0, NO_BYTES);
	memset(cpu_results, 0, NO_BYTES);

	clock_t cpu_start, cpu_end;
	cpu_start = clock();
	sum_array_cpu(h_a, h_b, cpu_results, size);
	cpu_end = clock();

	int* d_a, * d_b, * d_c;
	gpuErrchk(hipMalloc((int**)&d_a, NO_BYTES));
	gpuErrchk(hipMalloc((int**)&d_b, NO_BYTES));
	gpuErrchk(hipMalloc((int**)&d_c, NO_BYTES));

	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	clock_t mem_htod_start, mem_htod_end;
	mem_htod_start = clock();
	hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
	mem_htod_end = clock();

	clock_t gpu_start, gpu_end;
	gpu_start = clock();
	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, size);
	hipDeviceSynchronize();
	gpu_end = clock();

	clock_t dtoh_start, dtoh_end;
	dtoh_start = clock();
	hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
	dtoh_end = clock();

	//array comparison
	compare_arrays(gpu_results, cpu_results, size);

	printf("Sum array CPU execution time : %4.6f \n",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

	printf("Sum array GPU execution time : %4.6f \n",
		(double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));

	printf("htod mem transfer time : %4.6f \n",
		(double)((double)(mem_htod_end - mem_htod_start) / CLOCKS_PER_SEC));

	printf("dtoh mem transfer time : %4.6f \n",
		(double)((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));

	printf("Sum array total execution time : %4.6f \n",
		(double)((double)(dtoh_end - mem_htod_start) / CLOCKS_PER_SEC));

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);

	free(gpu_results);
	free(h_a);
	free(h_b);

	hipDeviceReset();
	return 0;
}
